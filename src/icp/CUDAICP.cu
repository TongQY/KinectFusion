#include "hip/hip_runtime.h"
#include "CUDAICPWrapper.cuh"

__global__ void fillSystem(
    float *sourcePts, float* targetPts, float *targetNormals,
    int n, float *A, float*b) {

    
    printf("AXAXAXAXAXA");
}

namespace CUDA {
	void createEquations(
        const float *sourcePts,
        const float *targetPts,
        const float *targetNrmls,
        int n,
        float *A,
        float *b) {


        float *dSourcePts, *dTargetPts, *dTargetNrmls;
        float *dA, *db;
        size_t size = sizeof(float);

        // allocate memory on the device
        hipMalloc((void**)&dSourcePts, size * n);
        hipMalloc((void**)&dTargetPts, size * n);
        hipMalloc((void**)&dTargetNrmls, size * n);
        hipMalloc((void**)&dA, size * n * 6);
        hipMalloc((void**)&db, size * n);

        // transfer data to the device
        hipMemcpy(
            dSourcePts, sourcePts, size * n, hipMemcpyHostToDevice);
        hipMemcpy(
            dTargetPts, targetPts, size * n, hipMemcpyHostToDevice);
        hipMemcpy(
            dTargetNrmls, targetNrmls, size * n, hipMemcpyHostToDevice);
        hipMemcpy(dA, A, size * n * 6, hipMemcpyHostToDevice);
        hipMemcpy(db, b, size * n, hipMemcpyHostToDevice);

        // execute the kernel
        fillSystem<<<1, 1>>>(
            dSourcePts, dTargetPts, dTargetNrmls, n, dA, db);

        // transfer data back
        hipMemcpy(A, dA, size * n * 6, hipMemcpyDeviceToHost);
        hipMemcpy(b, db, size * n, hipMemcpyDeviceToHost);

        // deallocate the device
        hipFree(dSourcePts);
        hipFree(dTargetPts);
        hipFree(dTargetNrmls);
        hipFree(dA);
        hipFree(db);
	}
}
