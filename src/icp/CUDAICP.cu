#include "hip/hip_runtime.h"
// Copyright 2020 Vladimir
// Author: Vladimir
#include <stdio.h>

#include "CUDAICPWrapper.cuh"

__global__ void fillSystem(
    float *x, float* y, float *n, int ptsNum, float *A, float*b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < ptsNum) {
        int id = 3 * i;
        A[6 * i] = n[id + 2] * x[id + 1] - n[id + 1] * x[id + 2];
        A[6 * i + 1] = n[id] * x[id + 2] - n[id + 2] * x[id];
        A[6 * i + 2] = n[id + 1] * x[id] - n[id] * x[id + 1];
        A[6 * i + 3] = n[id];
        A[6 * i + 4] = n[id + 1];
        A[6 * i + 5] = n[id + 2];
        b[i] = n[id] * y[id] + n[id + 1] * y[id + 1] +
            n[id + 2] * y[id + 2] - n[id] * x[id] -
            n[id + 1] * x[id + 1] - n[id + 2] * x[id + 2];
    }
}

namespace CUDA {
    void createEquations(
        const float *sourcePts,
        const float *targetPts,
        const float *targetNrmls,
        int n, float *A, float *b) {
        float *dSourcePts, *dTargetPts, *dTargetNrmls;
        float *dA, *db;
        size_t size = sizeof(float);

        // allocate memory on the device
        hipMalloc(&dSourcePts, 3 * size * n);
        hipMalloc(&dTargetPts, 3 * size * n);
        hipMalloc(&dTargetNrmls, 3 * size * n);
        hipMalloc(&dA, 6 * size * n);
        hipMalloc(&db, size * n);

        // transfer data to the device
        hipMemcpy(
            dSourcePts, sourcePts, 3 * size * n, hipMemcpyHostToDevice);
        hipMemcpy(
            dTargetPts, targetPts, 3 * size * n, hipMemcpyHostToDevice);
        hipMemcpy(
            dTargetNrmls, targetNrmls, 3 * size * n, hipMemcpyHostToDevice);

        // execute the kernel
        int block_size = 256;
        int grid_size = ((n + block_size) / block_size);
        fillSystem<<<block_size, grid_size>>>(
            dSourcePts, dTargetPts, dTargetNrmls, n, dA, db);

        // transfer data back
        hipMemcpy(A, dA, size * n * 6, hipMemcpyDeviceToHost);
        hipMemcpy(b, db, size * n, hipMemcpyDeviceToHost);

        // deallocate the device
        hipFree(dSourcePts);
        hipFree(dTargetPts);
        hipFree(dTargetNrmls);
        hipFree(dA);
        hipFree(db);
    }
}  // namespace CUDA
