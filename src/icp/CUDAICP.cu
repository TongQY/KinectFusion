#include "hip/hip_runtime.h"
// Copyright 2020 Vladimir
// Author: Vladimir
#include <stdio.h>

#include "CUDAICPWrapper.cuh"

__global__
void fillSystem(
    float *x, float* y, float *n, int ptsNum, float *A, float*b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < ptsNum) {
        int id = 3 * i;
        A[6 * i] = n[id + 2] * x[id + 1] - n[id + 1] * x[id + 2];
        A[6 * i + 1] = n[id] * x[id + 2] - n[id + 2] * x[id];
        A[6 * i + 2] = n[id + 1] * x[id] - n[id] * x[id + 1];
        A[6 * i + 3] = n[id];
        A[6 * i + 4] = n[id + 1];
        A[6 * i + 5] = n[id + 2];
        b[i] = n[id] * y[id] + n[id + 1] * y[id + 1] +
            n[id + 2] * y[id + 2] - n[id] * x[id] -
            n[id + 1] * x[id + 1] - n[id + 2] * x[id + 2];
    }
}

__global__
void findCorr(Vector3f* prevV, int prevSize) {

    printf("VERTICES FROM CUDA");
}

namespace CUDA {

    int* findCorrespondences(
        const Frame &prevFrame,
        const Frame &curFrame,
        const Matrix4f &estimatedPose) {

        Vector3f* dPrevV;
        Vector3f* dPrevN;
        Vector3f* dCurV;
        Vector3f* dCurN;
        Matrix4f* dExtrinsics;
        Matrix3f* dRotation;
        size_t vSize = sizeof(Vector3f);

        hipMalloc(&dPrevV, vSize * prevFrame.getVertexCount());
        hipMalloc(&dPrevN, vSize * prevFrame.getVertexCount());
        hipMalloc(&dCurN, vSize * curFrame.getVertexCount());
        hipMalloc(&dCurV, vSize * curFrame.getVertexCount());
        hipMalloc(&dExtrinsics, sizeof(Matrix4f));
        hipMalloc(&dRotation, sizeof(Matrix3f));

        hipMemcpy(
            dPrevV,
            prevFrame.getVertices().data(),
            vSize * prevFrame.getVertexCount(), hipMemcpyHostToDevice);

        hipMemcpy(
            dPrevN,
            prevFrame.getNormals().data(),
            vSize * prevFrame.getVertexCount(), hipMemcpyHostToDevice);

        hipMemcpy(
            dCurV,
            prevFrame.getVertices().data(),
            vSize * curFrame.getVertexCount(), hipMemcpyHostToDevice);

        hipMemcpy(
            dCurN,
            prevFrame.getVertices().data(),
            vSize * curFrame.getVertexCount(), hipMemcpyHostToDevice);

        findCorr<<<1, 256>>>(
            dPrevV, prevFrame.getVertexCount());


        hipFree(dPrevV);
        hipFree(dPrevN);
        hipFree(dCurV);
        hipFree(dCurN);
        hipFree(dExtrinsics);
        hipFree(dRotation);
        return 0;

    }

    void createEquations(
        const float *sourcePts,
        const float *targetPts,
        const float *targetNrmls,
        int n, float *A, float *b) {

        float *dSourcePts, *dTargetPts, *dTargetNrmls;
        float *dA, *db;
        size_t size = sizeof(float);

        // allocate memory on the device
        hipMalloc(&dSourcePts, 3 * size * n);
        hipMalloc(&dTargetPts, 3 * size * n);
        hipMalloc(&dTargetNrmls, 3 * size * n);
        hipMalloc(&dA, 6 * size * n);
        hipMalloc(&db, size * n);

        // transfer data to the device
        hipMemcpy(
            dSourcePts, sourcePts, 3 * size * n, hipMemcpyHostToDevice);
        hipMemcpy(
            dTargetPts, targetPts, 3 * size * n, hipMemcpyHostToDevice);
        hipMemcpy(
            dTargetNrmls, targetNrmls, 3 * size * n, hipMemcpyHostToDevice);

        // execute the kernel
        int block_size = 256;
        int grid_size = ((n + block_size) / block_size);
        fillSystem<<<block_size, grid_size>>>(
            dSourcePts, dTargetPts, dTargetNrmls, n, dA, db);

        // transfer data back
        hipMemcpy(A, dA, size * n * 6, hipMemcpyDeviceToHost);
        hipMemcpy(b, db, size * n, hipMemcpyDeviceToHost);

        // deallocate the device
        hipFree(dSourcePts);
        hipFree(dTargetPts);
        hipFree(dTargetNrmls);
        hipFree(dA);
        hipFree(db);
    }
}  // namespace CUDA
